#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

double get_wall_time() {
  struct timeval time;
  if ( gettimeofday(&time,NULL) ) {
      //  Handle error
      return 0;
  }
  return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

double get_cpu_time() {
  return (double)clock() / CLOCKS_PER_SEC;
}

#include "../cuMPI/src/cuMPI_runtime.h"

int myRank;                 // cuMPI comm local ranks
int nRanks;                 // total cuMPI comm ranks
int localRank;              // CUDA device ID

ncclUniqueId id;            // NCCL Unique ID
cuMPI_Comm comm;            // cuMPI comm
hipStream_t defaultStream; // CUDA stream generated for each GPU
uint64_t hostHashs[10];     // host name hash in cuMPI
char hostname[1024];        // host name for identification in cuMPI

// test P2P SendRecv method
int main() {
  cuMPI_Init(NULL, NULL);
  
  const int count = (1L << 30);
  const long long data_bytes = count * sizeof(float);
  const int max_times = 1024;
  // const int verify_range = 100;

  float *h_send = (float *)malloc(data_bytes);
        // *h_recv = (float *)malloc(data_bytes);
  for (int i = 0; i < count; ++i) {
    h_send[i] = localRank;
  }

  float *d_send = NULL, *d_recv = NULL;
  CUDA_CHECK(hipMalloc(&d_send, data_bytes));
  CUDA_CHECK(hipMalloc(&d_recv, data_bytes));
  
  CUDA_CHECK(hipMemcpy(d_send, h_send, data_bytes, hipMemcpyHostToDevice));

  cuMPI_Status status;
  int peer = 1 - myRank;

  double t1 = get_wall_time();
  
  for (int times = 0; times < max_times; ++times) {
    cuMPI_Sendrecv(d_send, count, cuMPI_FLOAT, peer, 0, d_recv, count, cuMPI_FLOAT, localRank, 0, comm, &status);
    hipStreamSynchronize(defaultStream);
  }

  double t2 = get_wall_time();

  const int data_gibytes = (data_bytes >> 30);
  printf("Send & Recv NCCL tests\n");
  printf("Data Size Each Time:\t%12.6f GBytes\n", (double)data_gibytes);
  printf("Performed times count:\t    %d\n", max_times);
  printf("Total Time cost:\t%12.6f seconds\n", t2 - t1);
  printf("Average Time cost:\t%12.6f seconds\n", (t2 - t1)/(double)(max_times));
  printf("Average Bus width:\t%12.6f TBytes/s\n", (double)(max_times * data_gibytes / 1024)/(t2 - t1));

  CUDA_CHECK(hipFree(d_send));
  CUDA_CHECK(hipFree(d_recv));
  free(h_send);
  cuMPI_Finalize();
  
  // CUDA_CHECK(hipMemcpy(h_recv, d_recv, verify_range * sizeof(float), hipMemcpyDeviceToHost));
  
  // for (int i = 0; i < verify_range; ++i) {
  //   // printf("[ %d ] -> [ %d ]\n", (int)(h_send[i]), (int)(h_recv[i]));
  //   assert( (int)abs((int)(h_send[i] - h_recv[i])) == 1 );
  // }

  // printf("Data is verified OK\n");

  // free(h_recv);
  return 0;
}
