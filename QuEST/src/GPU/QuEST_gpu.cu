#include "hip/hip_runtime.h"
#include "QuEST_gpu_internal.h"

ArgMatrix2 argifyMatrix2(ComplexMatrix2 m) {    
  ArgMatrix2 a;
  a.r0c0.real=m.real[0][0]; a.r0c0.imag=m.imag[0][0];
  a.r0c1.real=m.real[0][1]; a.r0c1.imag=m.imag[0][1];
  a.r1c0.real=m.real[1][0]; a.r1c0.imag=m.imag[1][0];
  a.r1c1.real=m.real[1][1]; a.r1c1.imag=m.imag[1][1];
  return a;
}

ArgMatrix4 argifyMatrix4(ComplexMatrix4 m) {     
  ArgMatrix4 a;
  a.r0c0.real=m.real[0][0]; a.r0c0.imag=m.imag[0][0];
  a.r0c1.real=m.real[0][1]; a.r0c1.imag=m.imag[0][1];
  a.r0c2.real=m.real[0][2]; a.r0c2.imag=m.imag[0][2];
  a.r0c3.real=m.real[0][3]; a.r0c3.imag=m.imag[0][3];
  a.r1c0.real=m.real[1][0]; a.r1c0.imag=m.imag[1][0];
  a.r1c1.real=m.real[1][1]; a.r1c1.imag=m.imag[1][1];
  a.r1c2.real=m.real[1][2]; a.r1c2.imag=m.imag[1][2];
  a.r1c3.real=m.real[1][3]; a.r1c3.imag=m.imag[1][3];
  a.r2c0.real=m.real[2][0]; a.r2c0.imag=m.imag[2][0];
  a.r2c1.real=m.real[2][1]; a.r2c1.imag=m.imag[2][1];
  a.r2c2.real=m.real[2][2]; a.r2c2.imag=m.imag[2][2];
  a.r2c3.real=m.real[2][3]; a.r2c3.imag=m.imag[2][3];
  a.r3c0.real=m.real[3][0]; a.r3c0.imag=m.imag[3][0];
  a.r3c1.real=m.real[3][1]; a.r3c1.imag=m.imag[3][1];
  a.r3c2.real=m.real[3][2]; a.r3c2.imag=m.imag[3][2];
  a.r3c3.real=m.real[3][3]; a.r3c3.imag=m.imag[3][3];
  return a;
}

void swapDouble(qreal **a, qreal **b){
  qreal *temp;
  temp = *a;
  *a = *b;
  *b = temp;
}

#ifdef __cplusplus
extern "C" {
#endif

//
// Derived from old single GPU version
// i.e. compared with CPU version, these functions only exist
// in QuEST_cpu.c, not in QuEST_cpu_local.c, and there are no
// related `*Local`-suffix functions for them.
//

__global__ void statevec_initDebugStateKernel(
  long long int chunkSize, 
  long long int chunkId, 
  qreal *stateVecReal, 
  qreal *stateVecImag)
{

  long long int index;
  long long int indexOffset = chunkSize * chunkId;

  index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index>=chunkSize) return;

  stateVecReal[index] = ((indexOffset + index)*2.0)/10.0;
  stateVecImag[index] = ((indexOffset + index)*2.0+1.0)/10.0;
}

void statevec_initDebugState(Qureg qureg)
{
  // stage 1 done!

  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_initDebugStateKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
      qureg.numAmpsPerChunk,
      qureg.chunkId,
      qureg.stateVec.real, 
      qureg.stateVec.imag);
}


__global__ void statevec_initStateOfSingleQubitKernel(
  long long int chunkSize,
  long long int numChunks,
  long long int chunkId,
  qreal *stateVecReal, 
  qreal *stateVecImag, 
  int qubitId, 
  int outcome)
{

  long long int index;
  long long int stateVecSize = chunkSize*numChunks;

  index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index>=chunkSize) return;

  qreal normFactor = 1.0/sqrt((qreal)stateVecSize/2);

  int bit = extractBit(qubitId, index+chunkId*chunkSize);
  if (bit==outcome) {
      stateVecReal[index] = normFactor;
      stateVecImag[index] = 0.0;
  } else {
      stateVecReal[index] = 0.0;
      stateVecImag[index] = 0.0;
  }
}

void statevec_initStateOfSingleQubit(Qureg *qureg, int qubitId, int outcome)
{
  // stage 1 done!

  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(qureg->numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_initStateOfSingleQubitKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
    qureg->numAmpsPerChunk, 
    qureg->numChunks,
    qureg->chunkId,
    qureg->stateVec.real, 
    qureg->stateVec.imag, 
    qubitId, outcome);
}

__global__ void statevec_compareStatesKernel(
  long long int chunkSize,
  qreal& mq1Real, 
  qreal& mq2Real,
  qreal& mq1Imag,
  qreal& mq2Imag,
  qreal precision,
  int *flag)
{
  long long int index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index >= chunkSize) return ;
  if (absReal(mq1Real - mq2Real) > precision ||
      absReal(mq1Imag - mq2Imag) > precision)
    {
      *flag = 1;
      return ;
    }
}

int statevec_compareStates(Qureg mq1, Qureg mq2, qreal precision)
{
  // stage 1 done!

  long long int chunkSize = mq1.numAmpsPerChunk;

  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(chunkSize)/threadsPerCUDABlock);
  
  int *d_flag_ptr = (int*)mallocZeroVarInDevice(1);
  for (long long int i=0; i<chunkSize; i++){
    statevec_compareStatesKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
      chunkSize,
      mq1.stateVec.real[i],
      mq2.stateVec.real[i],
      mq1.stateVec.imag[i],
      mq2.stateVec.imag[i],
      precision,
      d_flag_ptr);
  }
  int h_flag;
  hipMemcpy(&h_flag, d_flag_ptr, 1, hipMemcpyDeviceToHost);
  return 1 - h_flag;
}


__global__ void statevec_phaseShiftByTermKernel(Qureg qureg, const int targetQubit, qreal cosAngle, qreal sinAngle) {
  // stage 1 done!

  // !only for single gpu
  // long long int sizeBlock, sizeHalfBlock, thisBlock;
  // long long int indexUp, indexLo;

  qreal stateRealLo, stateImagLo;
  long long int thisTask, exactTask; // exactTask is global rank for distributed gpu.
  // const long long int numTasks = qureg.numAmpsPerChunk >> 1; // !only for single gpu
  const long long int numTasks = qureg.numAmpsPerChunk;

  // distributed gpu
  const long long int sizeChunk = qureg.numAmpsPerChunk;
  const long long int chunkId = qureg.chunkId;

  /* yh comment */
  /* sizeHalfBlock & sizeBlock using binary count trick
      e.g. qubit num = 3, target qubit = 1 (id begin with 0)
      000, 001 the center bit (qubit1) is 0 occur continuously for 2 times
      010, 011 when the center bit is 1, the next related bit(#) will add sizeBlock(=4) to index
      100, 101
      #110, 111
  */
  // !only for single gpu
  // sizeHalfBlock = 1LL << targetQubit;
  // sizeBlock     = 2LL * sizeHalfBlock;

  qreal *stateVecReal = qureg.stateVec.real;
  qreal *stateVecImag = qureg.stateVec.imag;

  // thisTask = blockIdx.x*blockDim.x + threadIdx.x;
  // if (thisTask>=numTasks) return;
  // thisBlock   = thisTask / sizeHalfBlock;
  // indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
  // indexLo     = indexUp + sizeHalfBlock;
  
  thisTask = blockIdx.x*blockDim.x + threadIdx.x;
  if (thisTask>=numTasks) return;
  exactTask = thisTask + chunkId*sizeChunk;

  if ( extractBit(targetQubit, exactTask) ) {
      
      stateRealLo = stateVecReal[thisTask];
      stateImagLo = stateVecImag[thisTask];

      stateVecReal[thisTask] = cosAngle*stateRealLo - sinAngle*stateImagLo;
      stateVecImag[thisTask] = sinAngle*stateRealLo + cosAngle*stateImagLo;
  }
}

void statevec_phaseShiftByTerm(Qureg qureg, const int targetQubit, Complex term)
{   
  // stage 1 done!
  // chunkId done!
  
  qreal cosAngle = term.real;
  qreal sinAngle = term.imag;
  
  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  
  // CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
  CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_phaseShiftByTermKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, cosAngle, sinAngle);
}


__global__ void statevec_controlledPhaseShiftKernel(Qureg qureg, const int idQubit1, const int idQubit2, qreal cosAngle, qreal sinAngle)
{
  long long int index;
  long long int stateVecSize;
  int bit1, bit2;

  const long long int chunkSize = qureg.numAmpsPerChunk;
  const long long int chunkId=qureg.chunkId;
  
  qreal stateRealLo, stateImagLo;

  stateVecSize = qureg.numAmpsPerChunk;
  qreal *stateVecReal = qureg.stateVec.real;
  qreal *stateVecImag = qureg.stateVec.imag;

  index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index>=stateVecSize) return;

  bit1 = extractBit (idQubit1, index+chunkId*chunkSize);
  bit2 = extractBit (idQubit2, index+chunkId*chunkSize);
  if (bit1 && bit2) {
      stateRealLo = stateVecReal[index];
      stateImagLo = stateVecImag[index];
      
      stateVecReal[index] = cosAngle*stateRealLo - sinAngle*stateImagLo;
      stateVecImag[index] = sinAngle*stateRealLo + cosAngle*stateImagLo;
  }
}

void statevec_controlledPhaseShift(Qureg qureg, const int idQubit1, const int idQubit2, qreal angle)
{
  // stage 1 done!

  qreal cosAngle = cos(angle);
  qreal sinAngle = sin(angle);
  
  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_controlledPhaseShiftKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, idQubit1, idQubit2, cosAngle, sinAngle);
}


__global__ void statevec_multiControlledPhaseShiftKernel(Qureg qureg, long long int mask, qreal cosAngle, qreal sinAngle) {
  qreal stateRealLo, stateImagLo;
  long long int index;
  long long int stateVecSize;

  const long long int chunkSize=qureg.numAmpsPerChunk;
  const long long int chunkId=qureg.chunkId;
  
  stateVecSize = qureg.numAmpsPerChunk;
  qreal *stateVecReal = qureg.stateVec.real;
  qreal *stateVecImag = qureg.stateVec.imag;
  
  index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index>=stateVecSize) return;

  if (mask == (mask & (index+chunkId*chunkSize)) ){
      stateRealLo = stateVecReal[index];
      stateImagLo = stateVecImag[index];
      stateVecReal[index] = cosAngle*stateRealLo - sinAngle*stateImagLo;
      stateVecImag[index] = sinAngle*stateRealLo + cosAngle*stateImagLo;
  }
}

void statevec_multiControlledPhaseShift(Qureg qureg, int *controlQubits, int numControlQubits, qreal angle)
{
  // stage 1 done!

  qreal cosAngle = cos(angle);
  qreal sinAngle = sin(angle);

  long long int mask = getQubitBitMask(controlQubits, numControlQubits);
      
  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_multiControlledPhaseShiftKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask, cosAngle, sinAngle);
}


__global__ void statevec_multiRotateZKernel(Qureg qureg, long long int mask, qreal cosAngle, qreal sinAngle) {
  
  long long int stateVecSize = qureg.numAmpsPerChunk;
  long long int index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index>=stateVecSize) return;
  
  const long long int chunkSize=qureg.numAmpsPerChunk;
  const long long int chunkId=qureg.chunkId;
  
  qreal *stateVecReal = qureg.stateVec.real;
  qreal *stateVecImag = qureg.stateVec.imag;
  
  // odd-parity target qubits get fac_j = -1
  int fac = getBitMaskParity(mask & (index+chunkId*chunkSize))? -1 : 1;
  qreal stateReal = stateVecReal[index];
  qreal stateImag = stateVecImag[index];
  
  stateVecReal[index] = cosAngle*stateReal + fac * sinAngle*stateImag;
  stateVecImag[index] = - fac * sinAngle*stateReal + cosAngle*stateImag;  
}

void statevec_multiRotateZ(Qureg qureg, long long int mask, qreal angle)
{   
  // stage 1 done!

  qreal cosAngle = cos(angle/2.0);
  qreal sinAngle = sin(angle/2.0);
      
  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
  statevec_multiRotateZKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask, cosAngle, sinAngle);
}


__global__ void statevec_controlledPhaseFlipKernel(Qureg qureg, const int idQubit1, const int idQubit2)
{
    long long int index;
    long long int stateVecSize;
    int bit1, bit2;

    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    bit1 = extractBit (idQubit1, index+chunkId*chunkSize);
    bit2 = extractBit (idQubit2, index+chunkId*chunkSize);
    if (bit1 && bit2) {
        stateVecReal [index] = - stateVecReal [index];
        stateVecImag [index] = - stateVecImag [index];
    }
}

void statevec_controlledPhaseFlip(Qureg qureg, const int idQubit1, const int idQubit2)
{
    // stage 1 done!
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledPhaseFlipKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, idQubit1, idQubit2);
}


__global__ void statevec_multiControlledPhaseFlipKernel(Qureg qureg, long long int mask)
{
    long long int index;
    long long int stateVecSize;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    if (mask == (mask & index) ){
        stateVecReal [index] = - stateVecReal [index];
        stateVecImag [index] = - stateVecImag [index];
    }
}

void statevec_multiControlledPhaseFlip(Qureg qureg, int *controlQubits, int numControlQubits)
{
    int threadsPerCUDABlock, CUDABlocks;
    long long int mask = getQubitBitMask(controlQubits, numControlQubits);
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_multiControlledPhaseFlipKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask);
}


__global__ void statevec_setWeightedQuregKernel(Complex fac1, Qureg qureg1, Complex fac2, Qureg qureg2, Complex facOut, Qureg out) {

  long long int ampInd = blockIdx.x*blockDim.x + threadIdx.x;
  long long int numAmpsToVisit = qureg1.numAmpsPerChunk;
  if (ampInd >= numAmpsToVisit) return;

  qreal *vecRe1 = qureg1.stateVec.real;
  qreal *vecIm1 = qureg1.stateVec.imag;
  qreal *vecRe2 = qureg2.stateVec.real;
  qreal *vecIm2 = qureg2.stateVec.imag;
  qreal *vecReOut = out.stateVec.real;
  qreal *vecImOut = out.stateVec.imag;

  qreal facRe1 = fac1.real; 
  qreal facIm1 = fac1.imag;
  qreal facRe2 = fac2.real;
  qreal facIm2 = fac2.imag;
  qreal facReOut = facOut.real;
  qreal facImOut = facOut.imag;

  qreal re1,im1, re2,im2, reOut,imOut;
  long long int index = ampInd;

  re1 = vecRe1[index]; im1 = vecIm1[index];
  re2 = vecRe2[index]; im2 = vecIm2[index];
  reOut = vecReOut[index];
  imOut = vecImOut[index];

  vecReOut[index] = (facReOut*reOut - facImOut*imOut) + (facRe1*re1 - facIm1*im1) + (facRe2*re2 - facIm2*im2);
  vecImOut[index] = (facReOut*imOut + facImOut*reOut) + (facRe1*im1 + facIm1*re1) + (facRe2*im2 + facIm2*re2);
}

void statevec_setWeightedQureg(Complex fac1, Qureg qureg1, Complex fac2, Qureg qureg2, Complex facOut, Qureg out) {

  long long int numAmpsToVisit = qureg1.numAmpsPerChunk;

  int threadsPerCUDABlock, CUDABlocks;
  threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
  CUDABlocks = ceil(numAmpsToVisit / (qreal) threadsPerCUDABlock);
  statevec_setWeightedQuregKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
      fac1, qureg1, fac2, qureg2, facOut, out
  );
}



// densmatr
void densmatr_initPureState(Qureg targetQureg, Qureg copyQureg){}
void densmatr_initPlusState(Qureg qureg){}
void densmatr_initClassicalState(Qureg qureg, long long int stateInd){}
void densmatr_collapseToKnownProbOutcome(Qureg qureg, const int measureQubit, int outcome, qreal outcomeProb){}
void densmatr_mixDensityMatrix(Qureg combineQureg, qreal otherProb, Qureg otherQureg){}
void densmatr_oneQubitDegradeOffDiagonal(Qureg qureg, const int targetQubit, qreal dephFac){}
void densmatr_mixDephasing(Qureg qureg, const int targetQubit, qreal dephase){}
void densmatr_mixTwoQubitDephasing(Qureg qureg, int qubit1, int qubit2, qreal dephase){}
void densmatr_mixDepolarising(Qureg qureg, const int targetQubit, qreal depolLevel){}
void densmatr_mixDamping(Qureg qureg, const int targetQubit, qreal damping){}
void densmatr_mixTwoQubitDepolarising(Qureg qureg, int qubit1, int qubit2, qreal depolLevel){}
qreal densmatr_calcFidelity(Qureg qureg, Qureg pureState){return (qreal)0;}
qreal densmatr_calcHilbertSchmidtDistanceSquared(Qureg a, Qureg b){return (qreal)0;}
qreal densmatr_calcPurity(Qureg qureg){return (qreal)0;}
qreal densmatr_calcProbOfOutcome(Qureg qureg, const int measureQubit, int outcome){return (qreal)0;}
qreal densmatr_findProbabilityOfZero(Qureg qureg, const int measureQubit){return (qreal)0;}
qreal densmatr_calcTotalProb(Qureg qureg){return (qreal)0;}
qreal densmatr_calcHilbertSchmidtDistance(Qureg a, Qureg b){return (qreal)0;}
qreal densmatr_calcInnerProduct(Qureg a, Qureg b){return (qreal)0;}


#ifdef __cplusplus
}
#endif
