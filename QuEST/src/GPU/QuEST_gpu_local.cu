#include "hip/hip_runtime.h"
#include "QuEST_gpu_internal.h"

/*
 * state vector and density matrix operations 
 */

#ifdef __cplusplus
extern "C" {
#endif

qreal statevec_getRealAmpLocal(Qureg qureg, long long int index){
    // stage 1 done!
    qreal el=0;
    hipMemcpy(&el, &(qureg.stateVec.real[index]), 
            sizeof(*(qureg.stateVec.real)), hipMemcpyDeviceToHost);
    return el;
}

qreal statevec_getImagAmpLocal(Qureg qureg, long long int index){
    // stage 1 done!
    qreal el=0;
    hipMemcpy(&el, &(qureg.stateVec.imag[index]), 
            sizeof(*(qureg.stateVec.imag)), hipMemcpyDeviceToHost);
    return el;
}

__global__ void statevec_initDebugStateKernel(long long int stateVecSize, qreal *stateVecReal, qreal *stateVecImag){
    long long int index;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    stateVecReal[index] = (index*2.0)/10.0;
    stateVecImag[index] = (index*2.0+1.0)/10.0;
}

void statevec_initDebugState(Qureg qureg)
{
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_initDebugStateKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
        qureg.numAmpsPerChunk,
        qureg.stateVec.real, 
        qureg.stateVec.imag);
}

__global__ void statevec_initStateOfSingleQubitKernel(long long int stateVecSize, qreal *stateVecReal, qreal *stateVecImag, int qubitId, int outcome){
    long long int index;
    int bit;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    qreal normFactor = 1.0/sqrt((qreal)stateVecSize/2);
    bit = extractBit(qubitId, index);
    if (bit==outcome) {
        stateVecReal[index] = normFactor;
        stateVecImag[index] = 0.0;
    } else {
        stateVecReal[index] = 0.0;
        stateVecImag[index] = 0.0;
    }
}

void statevec_initStateOfSingleQubit(Qureg *qureg, int qubitId, int outcome)
{
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg->numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_initStateOfSingleQubitKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg->numAmpsPerChunk, qureg->stateVec.real, qureg->stateVec.imag, qubitId, outcome);
}

int statevec_compareStates(Qureg mq1, Qureg mq2, qreal precision){
    qreal diff;
    int chunkSize = mq1.numAmpsPerChunk;

    copyStateFromGPU(mq1);
    copyStateFromGPU(mq2);

    for (int i=0; i<chunkSize; i++){
        diff = mq1.stateVec.real[i] - mq2.stateVec.real[i];
        if (diff<0) diff *= -1;
        if (diff>precision) return 0;
        diff = mq1.stateVec.imag[i] - mq2.stateVec.imag[i];
        if (diff<0) diff *= -1;
        if (diff>precision) return 0;
    }
    return 1;
}

__global__ void statevec_compactUnitaryKernel (Qureg qureg, const int rotQubit, Complex alpha, Complex beta){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;

    sizeHalfBlock = 1LL << rotQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;
    qreal alphaImag=alpha.imag, alphaReal=alpha.real;
    qreal betaImag=beta.imag, betaReal=beta.real;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    // store current state vector values in temp variables
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    stateRealLo = stateVecReal[indexLo];
    stateImagLo = stateVecImag[indexLo];

    // state[indexUp] = alpha * state[indexUp] - conj(beta)  * state[indexLo]
    stateVecReal[indexUp] = alphaReal*stateRealUp - alphaImag*stateImagUp 
        - betaReal*stateRealLo - betaImag*stateImagLo;
    stateVecImag[indexUp] = alphaReal*stateImagUp + alphaImag*stateRealUp 
        - betaReal*stateImagLo + betaImag*stateRealLo;

    // state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
    stateVecReal[indexLo] = betaReal*stateRealUp - betaImag*stateImagUp 
        + alphaReal*stateRealLo + alphaImag*stateImagLo;
    stateVecImag[indexLo] = betaReal*stateImagUp + betaImag*stateRealUp 
        + alphaReal*stateImagLo - alphaImag*stateRealLo;
}

void statevec_compactUnitaryLocal(Qureg qureg, const int targetQubit, Complex alpha, Complex beta) 
{
    // stage 1 done!
    // chunkID done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_compactUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, alpha, beta);
}

__global__ void statevec_controlledCompactUnitaryKernel (Qureg qureg, const int controlQubit, const int targetQubit, Complex alpha, Complex beta){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;
    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;
    int controlBit;

    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;
    qreal alphaImag=alpha.imag, alphaReal=alpha.real;
    qreal betaImag=beta.imag, betaReal=beta.real;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    controlBit = extractBit(controlQubit, indexUp+chunkId*chunkSize);
    if (controlBit){
        // store current state vector values in temp variables
        stateRealUp = stateVecReal[indexUp];
        stateImagUp = stateVecImag[indexUp];

        stateRealLo = stateVecReal[indexLo];
        stateImagLo = stateVecImag[indexLo];

        // state[indexUp] = alpha * state[indexUp] - conj(beta)  * state[indexLo]
        stateVecReal[indexUp] = alphaReal*stateRealUp - alphaImag*stateImagUp 
            - betaReal*stateRealLo - betaImag*stateImagLo;
        stateVecImag[indexUp] = alphaReal*stateImagUp + alphaImag*stateRealUp 
            - betaReal*stateImagLo + betaImag*stateRealLo;

        // state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
        stateVecReal[indexLo] = betaReal*stateRealUp - betaImag*stateImagUp 
            + alphaReal*stateRealLo + alphaImag*stateImagLo;
        stateVecImag[indexLo] = betaReal*stateImagUp + betaImag*stateRealUp 
            + alphaReal*stateImagLo - alphaImag*stateRealLo;
    }
}

void statevec_controlledCompactUnitaryLocal(Qureg qureg, const int controlQubit, const int targetQubit, Complex alpha, Complex beta) 
{
    // stage 1 done!
    // chunkID done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_controlledCompactUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit, alpha, beta);
}

__global__ void statevec_unitaryKernel(Qureg qureg, const int targetQubit, ArgMatrix2 u){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;

    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    // store current state vector values in temp variables
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    stateRealLo = stateVecReal[indexLo];
    stateImagLo = stateVecImag[indexLo];

    // state[indexUp] = u00 * state[indexUp] + u01 * state[indexLo]
    stateVecReal[indexUp] = u.r0c0.real*stateRealUp - u.r0c0.imag*stateImagUp 
        + u.r0c1.real*stateRealLo - u.r0c1.imag*stateImagLo;
    stateVecImag[indexUp] = u.r0c0.real*stateImagUp + u.r0c0.imag*stateRealUp 
        + u.r0c1.real*stateImagLo + u.r0c1.imag*stateRealLo;

    // state[indexLo] = u10  * state[indexUp] + u11 * state[indexLo]
    stateVecReal[indexLo] = u.r1c0.real*stateRealUp  - u.r1c0.imag*stateImagUp 
        + u.r1c1.real*stateRealLo  -  u.r1c1.imag*stateImagLo;
    stateVecImag[indexLo] = u.r1c0.real*stateImagUp + u.r1c0.imag*stateRealUp 
        + u.r1c1.real*stateImagLo + u.r1c1.imag*stateRealLo;
}

void statevec_unitaryLocal(Qureg qureg, const int targetQubit, ComplexMatrix2 u)
{
    // stage 1 done!
    // chunkId done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_unitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, argifyMatrix2(u));
}

__global__ void statevec_multiControlledMultiQubitUnitaryKernel(
    Qureg qureg, long long int ctrlMask, int* targs, int numTargs, 
    qreal* uRe, qreal* uIm, long long int* ampInds, qreal* reAmps, qreal* imAmps, long long int numTargAmps)
{
    
    // decide the amplitudes this thread will modify
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;                        
    long long int numTasks = qureg.numAmpsPerChunk >> numTargs; // kernel called on every 1 in 2^numTargs amplitudes
    if (thisTask>=numTasks) return;
    
    // find this task's start index (where all targs are 0)
    long long int ind00 = insertZeroBits(thisTask, targs, numTargs);
    
    // this task only modifies amplitudes if control qubits are 1 for this state
    if (ctrlMask && (ctrlMask&ind00) != ctrlMask)
        return;
        
    qreal *reVec = qureg.stateVec.real;
    qreal *imVec = qureg.stateVec.imag;
    
    /*
    each thread needs:
        long long int ampInds[numAmps];
        qreal reAmps[numAmps];
        qreal imAmps[numAmps];
    but instead has access to shared arrays, with below stride and offset
    */
    size_t stride = gridDim.x*blockDim.x;
    size_t offset = blockIdx.x*blockDim.x + threadIdx.x;
    
    // determine the indices and record values of target amps
    long long int ind;
    for (int i=0; i < numTargAmps; i++) {
        
        // get global index of current target qubit assignment
        ind = ind00;
        for (int t=0; t < numTargs; t++)
            if (extractBit(t, i))
                ind = flipBit(ind, targs[t]);
        
        ampInds[i*stride+offset] = ind;
        reAmps [i*stride+offset] = reVec[ind];
        imAmps [i*stride+offset] = imVec[ind];
    }
    
    // update the amplitudes
    for (int r=0; r < numTargAmps; r++) {
        ind = ampInds[r*stride+offset];
        reVec[ind] = 0;
        imVec[ind] = 0;
        for (int c=0; c < numTargAmps; c++) {
            qreal uReElem = uRe[c + r*numTargAmps];
            qreal uImElem = uIm[c + r*numTargAmps];
            reVec[ind] += reAmps[c*stride+offset]*uReElem - imAmps[c*stride+offset]*uImElem;
            imVec[ind] += reAmps[c*stride+offset]*uImElem + imAmps[c*stride+offset]*uReElem;
        }
    }
}

void statevec_multiControlledMultiQubitUnitaryLocal(Qureg qureg, long long int ctrlMask, int* targs, const int numTargs, ComplexMatrixN u)
{
    int threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>numTargs)/threadsPerCUDABlock);
    
    // allocate device space for global {targs} (length: numTargs) and populate
    int *d_targs;
    size_t targMemSize = numTargs * sizeof *d_targs;
    hipMalloc(&d_targs, targMemSize);
    hipMemcpy(d_targs, targs, targMemSize, hipMemcpyHostToDevice);
    
    // flatten out the u.real and u.imag lists
    int uNumRows = (1 << u.numQubits);
    qreal* uReFlat = (qreal*) malloc(uNumRows*uNumRows * sizeof *uReFlat);
    qreal* uImFlat = (qreal*) malloc(uNumRows*uNumRows * sizeof *uImFlat);
    long long int i = 0;
    for (int r=0; r < uNumRows; r++)
        for (int c=0; c < uNumRows; c++) {
            uReFlat[i] = u.real[r][c];
            uImFlat[i] = u.imag[r][c];
            i++;
        }
    
    // allocate device space for global u.real and u.imag (flatten by concatenating rows) and populate
    qreal* d_uRe;
    qreal* d_uIm;
    size_t uMemSize = uNumRows*uNumRows * sizeof *d_uRe; // size of each of d_uRe and d_uIm
    hipMalloc(&d_uRe, uMemSize);
    hipMalloc(&d_uIm, uMemSize);
    hipMemcpy(d_uRe, uReFlat, uMemSize, hipMemcpyHostToDevice);
    hipMemcpy(d_uIm, uImFlat, uMemSize, hipMemcpyHostToDevice);
    
    // allocate device Wspace for thread-local {ampInds}, {reAmps}, {imAmps} (length: 1<<numTargs)
    long long int *d_ampInds;
    qreal *d_reAmps;
    qreal *d_imAmps;
    size_t gridSize = (size_t) threadsPerCUDABlock * CUDABlocks;
    int numTargAmps = uNumRows;
    hipMalloc(&d_ampInds, numTargAmps*gridSize * sizeof *d_ampInds);
    hipMalloc(&d_reAmps,  numTargAmps*gridSize * sizeof *d_reAmps);
    hipMalloc(&d_imAmps,  numTargAmps*gridSize * sizeof *d_imAmps);
    
    // call kernel
    statevec_multiControlledMultiQubitUnitaryKernel<<<CUDABlocks,threadsPerCUDABlock>>>(
        qureg, ctrlMask, d_targs, numTargs, d_uRe, d_uIm, d_ampInds, d_reAmps, d_imAmps, numTargAmps);
        
    // free kernel memory
    free(uReFlat);
    free(uImFlat);
    hipFree(d_targs);
    hipFree(d_uRe);
    hipFree(d_uIm);
    hipFree(d_ampInds);
    hipFree(d_reAmps);
    hipFree(d_imAmps);
}

__global__ void statevec_multiControlledTwoQubitUnitaryKernel(Qureg qureg, long long int ctrlMask, const int q1, const int q2, ArgMatrix4 u){
    
    // decide the 4 amplitudes this thread will modify
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;                        
    long long int numTasks = qureg.numAmpsPerChunk >> 2; // kernel called on every 1 in 4 amplitudes
    if (thisTask>=numTasks) return;
    
    qreal *reVec = qureg.stateVec.real;
    qreal *imVec = qureg.stateVec.imag;
    
    // find indices of amplitudes to modify (treat q1 as the least significant bit)
    long long int ind00, ind01, ind10, ind11;
    ind00 = insertTwoZeroBits(thisTask, q1, q2);
    
    // modify only if control qubits are 1 for this state
    if (ctrlMask && (ctrlMask&ind00) != ctrlMask)
        return;
    
    ind01 = flipBit(ind00, q1);
    ind10 = flipBit(ind00, q2);
    ind11 = flipBit(ind01, q2);
    
    // extract statevec amplitudes 
    qreal re00, re01, re10, re11;
    qreal im00, im01, im10, im11;
    re00 = reVec[ind00]; im00 = imVec[ind00];
    re01 = reVec[ind01]; im01 = imVec[ind01];
    re10 = reVec[ind10]; im10 = imVec[ind10];
    re11 = reVec[ind11]; im11 = imVec[ind11];
    
    // apply u * {amp00, amp01, amp10, amp11}
    reVec[ind00] = 
        u.r0c0.real*re00 - u.r0c0.imag*im00 +
        u.r0c1.real*re01 - u.r0c1.imag*im01 +
        u.r0c2.real*re10 - u.r0c2.imag*im10 +
        u.r0c3.real*re11 - u.r0c3.imag*im11;
    imVec[ind00] =
        u.r0c0.imag*re00 + u.r0c0.real*im00 +
        u.r0c1.imag*re01 + u.r0c1.real*im01 +
        u.r0c2.imag*re10 + u.r0c2.real*im10 +
        u.r0c3.imag*re11 + u.r0c3.real*im11;
        
    reVec[ind01] = 
        u.r1c0.real*re00 - u.r1c0.imag*im00 +
        u.r1c1.real*re01 - u.r1c1.imag*im01 +
        u.r1c2.real*re10 - u.r1c2.imag*im10 +
        u.r1c3.real*re11 - u.r1c3.imag*im11;
    imVec[ind01] =
        u.r1c0.imag*re00 + u.r1c0.real*im00 +
        u.r1c1.imag*re01 + u.r1c1.real*im01 +
        u.r1c2.imag*re10 + u.r1c2.real*im10 +
        u.r1c3.imag*re11 + u.r1c3.real*im11;
        
    reVec[ind10] = 
        u.r2c0.real*re00 - u.r2c0.imag*im00 +
        u.r2c1.real*re01 - u.r2c1.imag*im01 +
        u.r2c2.real*re10 - u.r2c2.imag*im10 +
        u.r2c3.real*re11 - u.r2c3.imag*im11;
    imVec[ind10] =
        u.r2c0.imag*re00 + u.r2c0.real*im00 +
        u.r2c1.imag*re01 + u.r2c1.real*im01 +
        u.r2c2.imag*re10 + u.r2c2.real*im10 +
        u.r2c3.imag*re11 + u.r2c3.real*im11;    
        
    reVec[ind11] = 
        u.r3c0.real*re00 - u.r3c0.imag*im00 +
        u.r3c1.real*re01 - u.r3c1.imag*im01 +
        u.r3c2.real*re10 - u.r3c2.imag*im10 +
        u.r3c3.real*re11 - u.r3c3.imag*im11;
    imVec[ind11] =
        u.r3c0.imag*re00 + u.r3c0.real*im00 +
        u.r3c1.imag*re01 + u.r3c1.real*im01 +
        u.r3c2.imag*re10 + u.r3c2.real*im10 +
        u.r3c3.imag*re11 + u.r3c3.real*im11;    
}

void statevec_multiControlledTwoQubitUnitaryLocal(Qureg qureg, long long int ctrlMask, const int q1, const int q2, ComplexMatrix4 u)
{
    int threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>2)/threadsPerCUDABlock); // one kernel eval for every 4 amplitudes
    statevec_multiControlledTwoQubitUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, ctrlMask, q1, q2, argifyMatrix4(u));
}

__global__ void statevec_controlledUnitaryKernel(Qureg qureg, const int controlQubit, const int targetQubit, ArgMatrix2 u){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;
    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;

    int controlBit;

    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    // store current state vector values in temp variables
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    stateRealLo = stateVecReal[indexLo];
    stateImagLo = stateVecImag[indexLo];

    controlBit = extractBit(controlQubit, indexUp+chunkId*chunkSize);
    if (controlBit){
        // state[indexUp] = u00 * state[indexUp] + u01 * state[indexLo]
        stateVecReal[indexUp] = u.r0c0.real*stateRealUp - u.r0c0.imag*stateImagUp 
            + u.r0c1.real*stateRealLo - u.r0c1.imag*stateImagLo;
        stateVecImag[indexUp] = u.r0c0.real*stateImagUp + u.r0c0.imag*stateRealUp 
            + u.r0c1.real*stateImagLo + u.r0c1.imag*stateRealLo;

        // state[indexLo] = u10  * state[indexUp] + u11 * state[indexLo]
        stateVecReal[indexLo] = u.r1c0.real*stateRealUp  - u.r1c0.imag*stateImagUp 
            + u.r1c1.real*stateRealLo  -  u.r1c1.imag*stateImagLo;
        stateVecImag[indexLo] = u.r1c0.real*stateImagUp + u.r1c0.imag*stateRealUp 
            + u.r1c1.real*stateImagLo + u.r1c1.imag*stateRealLo;
    }
}

void statevec_controlledUnitaryLocal(Qureg qureg, const int controlQubit, const int targetQubit, ComplexMatrix2 u)
{
    // stage 1 done!
    // chunkId done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_controlledUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit, argifyMatrix2(u));
}

__global__ void statevec_multiControlledUnitaryKernel(
    Qureg qureg, 
    long long int ctrlQubitsMask, long long int ctrlFlipMask, 
    const int targetQubit, ArgMatrix2 u
){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;
    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;


    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    if (ctrlQubitsMask == (ctrlQubitsMask & ((indexUp+chunkId*chunkSize) ^ ctrlFlipMask))) {
        // store current state vector values in temp variables
        stateRealUp = stateVecReal[indexUp];
        stateImagUp = stateVecImag[indexUp];

        stateRealLo = stateVecReal[indexLo];
        stateImagLo = stateVecImag[indexLo];

        // state[indexUp] = u00 * state[indexUp] + u01 * state[indexLo]
        stateVecReal[indexUp] = u.r0c0.real*stateRealUp - u.r0c0.imag*stateImagUp 
            + u.r0c1.real*stateRealLo - u.r0c1.imag*stateImagLo;
        stateVecImag[indexUp] = u.r0c0.real*stateImagUp + u.r0c0.imag*stateRealUp 
            + u.r0c1.real*stateImagLo + u.r0c1.imag*stateRealLo;

        // state[indexLo] = u10  * state[indexUp] + u11 * state[indexLo]
        stateVecReal[indexLo] = u.r1c0.real*stateRealUp  - u.r1c0.imag*stateImagUp 
            + u.r1c1.real*stateRealLo  -  u.r1c1.imag*stateImagLo;
        stateVecImag[indexLo] = u.r1c0.real*stateImagUp + u.r1c0.imag*stateRealUp 
            + u.r1c1.real*stateImagLo + u.r1c1.imag*stateRealLo;
    }
}

void statevec_multiControlledUnitaryLocal(
    Qureg qureg, 
    long long int ctrlQubitsMask, long long int ctrlFlipMask, 
    const int targetQubit, ComplexMatrix2 u
){
    // stage 1 done!
    // chunkId done!
    
    int threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_multiControlledUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
        qureg, ctrlQubitsMask, ctrlFlipMask, targetQubit, argifyMatrix2(u));
}

__global__ void statevec_pauliXKernel(Qureg qureg, const int targetQubit){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,                             // storage for previous state values
           stateImagUp;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;

    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    // store current state vector values in temp variables
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    stateVecReal[indexUp] = stateVecReal[indexLo];
    stateVecImag[indexUp] = stateVecImag[indexLo];

    stateVecReal[indexLo] = stateRealUp;
    stateVecImag[indexLo] = stateImagUp;
}

void statevec_pauliXLocal(Qureg qureg, const int targetQubit) 
{
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_pauliXKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit);
}

__global__ void statevec_pauliYKernel(Qureg qureg, const int targetQubit, const int conjFac){

    long long int sizeHalfBlock = 1LL << targetQubit;
    long long int sizeBlock     = 2LL * sizeHalfBlock;
    long long int numTasks      = qureg.numAmpsPerChunk >> 1;
    long long int thisTask      = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;
    
    long long int thisBlock     = thisTask / sizeHalfBlock;
    long long int indexUp       = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    long long int indexLo       = indexUp + sizeHalfBlock;
    qreal  stateRealUp, stateImagUp;

    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    // update under +-{{0, -i}, {i, 0}}
    stateVecReal[indexUp] = conjFac * stateVecImag[indexLo];
    stateVecImag[indexUp] = conjFac * -stateVecReal[indexLo];
    stateVecReal[indexLo] = conjFac * -stateImagUp;
    stateVecImag[indexLo] = conjFac * stateRealUp;
}

void statevec_pauliYLocal(Qureg qureg, const int targetQubit) 
{
    // chunkID done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_pauliYKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, 1);
}

void statevec_pauliYConjLocal(Qureg qureg, const int targetQubit) 
{
    // chunkID done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_pauliYKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, -1);
}

__global__ void statevec_controlledPauliYKernel(Qureg qureg, const int controlQubit, const int targetQubit, const int conjFac)
{
    long long int index;
    long long int sizeBlock, sizeHalfBlock;
    long long int stateVecSize;
    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;
    int controlBit;

    qreal   stateRealUp, stateImagUp; 
    long long int thisBlock, indexUp, indexLo;                                     
    sizeHalfBlock = 1LL << targetQubit;
    sizeBlock     = 2LL * sizeHalfBlock;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=(stateVecSize>>1)) return;
    thisBlock   = index / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + index%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    controlBit = extractBit(controlQubit, indexUp+chunkId*chunkSize);
    if (controlBit){

        stateRealUp = stateVecReal[indexUp];
        stateImagUp = stateVecImag[indexUp];

        // update under +-{{0, -i}, {i, 0}}
        stateVecReal[indexUp] = conjFac * stateVecImag[indexLo];
        stateVecImag[indexUp] = conjFac * -stateVecReal[indexLo];
        stateVecReal[indexLo] = conjFac * -stateImagUp;
        stateVecImag[indexLo] = conjFac * stateRealUp;
    }
}

void statevec_controlledPauliYLocal(Qureg qureg, const int controlQubit, const int targetQubit)
{
    // chunkID done!

    int conjFactor = 1;
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledPauliYKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit, conjFactor);
}

void statevec_controlledPauliYConjLocal(Qureg qureg, const int controlQubit, const int targetQubit)
{
    // chunkID done!

    int conjFactor = -1;
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledPauliYKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit, conjFactor);
}

__global__ void statevec_phaseShiftByTermKernel(Qureg qureg, const int targetQubit, qreal cosAngle, qreal sinAngle) {
    // stage 1 done!

    // !only for single gpu
    // long long int sizeBlock, sizeHalfBlock, thisBlock;
    // long long int indexUp, indexLo;

    qreal stateRealLo, stateImagLo;
    long long int thisTask, exactTask; // exactTask is global rank for distributed gpu.
    // const long long int numTasks = qureg.numAmpsPerChunk >> 1; // !only for single gpu
    const long long int numTasks = qureg.numAmpsPerChunk;

    // distributed gpu
    const long long int sizeChunk = qureg.numAmpsPerChunk;
    const long long int chunkId = qureg.chunkId;

    /* yh comment */
    /* sizeHalfBlock & sizeBlock using binary count trick
        e.g. qubit num = 3, target qubit = 1 (id begin with 0)
        000, 001 the center bit (qubit1) is 0 occur continuously for 2 times
        010, 011 when the center bit is 1, the next related bit(#) will add sizeBlock(=4) to index
        100, 101
        #110, 111
    */
    // !only for single gpu
    // sizeHalfBlock = 1LL << targetQubit;
    // sizeBlock     = 2LL * sizeHalfBlock;

    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    // thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    // if (thisTask>=numTasks) return;
    // thisBlock   = thisTask / sizeHalfBlock;
    // indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    // indexLo     = indexUp + sizeHalfBlock;
    
    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;
    exactTask = thisTask + chunkId*sizeChunk;

    if ( extractBit(targetQubit, exactTask) ) {
        
        stateRealLo = stateVecReal[thisTask];
        stateImagLo = stateVecImag[thisTask];

        stateVecReal[thisTask] = cosAngle*stateRealLo - sinAngle*stateImagLo;
        stateVecImag[thisTask] = sinAngle*stateRealLo + cosAngle*stateImagLo;
    }
}

void statevec_phaseShiftByTerm(Qureg qureg, const int targetQubit, Complex term)
{   
    // stage 1 done!
    qreal cosAngle = term.real;
    qreal sinAngle = term.imag;
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    
    // CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_phaseShiftByTermKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit, cosAngle, sinAngle);
}

__global__ void statevec_controlledPhaseShiftKernel(Qureg qureg, const int idQubit1, const int idQubit2, qreal cosAngle, qreal sinAngle)
{
    long long int index;
    long long int stateVecSize;
    int bit1, bit2;
    qreal stateRealLo, stateImagLo;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    bit1 = extractBit (idQubit1, index);
    bit2 = extractBit (idQubit2, index);
    if (bit1 && bit2) {
        stateRealLo = stateVecReal[index];
        stateImagLo = stateVecImag[index];
        
        stateVecReal[index] = cosAngle*stateRealLo - sinAngle*stateImagLo;
        stateVecImag[index] = sinAngle*stateRealLo + cosAngle*stateImagLo;
    }
}

void statevec_controlledPhaseShift(Qureg qureg, const int idQubit1, const int idQubit2, qreal angle)
{
    qreal cosAngle = cos(angle);
    qreal sinAngle = sin(angle);
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledPhaseShiftKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, idQubit1, idQubit2, cosAngle, sinAngle);
}

__global__ void statevec_multiControlledPhaseShiftKernel(Qureg qureg, long long int mask, qreal cosAngle, qreal sinAngle) {
    qreal stateRealLo, stateImagLo;
    long long int index;
    long long int stateVecSize;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;
    
    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    if (mask == (mask & index) ){
        stateRealLo = stateVecReal[index];
        stateImagLo = stateVecImag[index];
        stateVecReal[index] = cosAngle*stateRealLo - sinAngle*stateImagLo;
        stateVecImag[index] = sinAngle*stateRealLo + cosAngle*stateImagLo;
    }
}

void statevec_multiControlledPhaseShift(Qureg qureg, int *controlQubits, int numControlQubits, qreal angle)
{   
    qreal cosAngle = cos(angle);
    qreal sinAngle = sin(angle);

    long long int mask = getQubitBitMask(controlQubits, numControlQubits);
        
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_multiControlledPhaseShiftKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask, cosAngle, sinAngle);
}

__global__ void statevec_multiRotateZKernel(Qureg qureg, long long int mask, qreal cosAngle, qreal sinAngle) {
    
    long long int stateVecSize = qureg.numAmpsPerChunk;
    long long int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;
    
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;
    
    int fac = getBitMaskParity(mask & index)? -1 : 1;
    qreal stateReal = stateVecReal[index];
    qreal stateImag = stateVecImag[index];
    
    stateVecReal[index] = cosAngle*stateReal + fac * sinAngle*stateImag;
    stateVecImag[index] = - fac * sinAngle*stateReal + cosAngle*stateImag;  
}

void statevec_multiRotateZ(Qureg qureg, long long int mask, qreal angle)
{   
    qreal cosAngle = cos(angle/2.0);
    qreal sinAngle = sin(angle/2.0);
        
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_multiRotateZKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask, cosAngle, sinAngle);
}


qreal statevec_calcTotalProbLocal(Qureg qureg){
    /* IJB - implemented using Kahan summation for greater accuracy at a slight floating
       point operation overhead. For more details see https://en.wikipedia.org/wiki/Kahan_summation_algorithm */
    /* Don't change the bracketing in this routine! */
    qreal pTotal=0;
    qreal y, t, c;
    long long int index;
    long long int numAmpsPerRank = qureg.numAmpsPerChunk;

    copyStateFromGPU(qureg);

    c = 0.0;
    for (index=0; index<numAmpsPerRank; index++){
        /* Perform pTotal+=qureg.stateVec.real[index]*qureg.stateVec.real[index]; by Kahan */
        // pTotal+=qureg.stateVec.real[index]*qureg.stateVec.real[index];
        y = qureg.stateVec.real[index]*qureg.stateVec.real[index] - c;
        t = pTotal + y;
        c = ( t - pTotal ) - y;
        pTotal = t;

        /* Perform pTotal+=qureg.stateVec.imag[index]*qureg.stateVec.imag[index]; by Kahan */
        //pTotal+=qureg.stateVec.imag[index]*qureg.stateVec.imag[index];
        y = qureg.stateVec.imag[index]*qureg.stateVec.imag[index] - c;
        t = pTotal + y;
        c = ( t - pTotal ) - y;
        pTotal = t;


    }
    return pTotal;
}

__global__ void statevec_controlledPhaseFlipKernel(Qureg qureg, const int idQubit1, const int idQubit2)
{
    long long int index;
    long long int stateVecSize;
    int bit1, bit2;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    bit1 = extractBit (idQubit1, index);
    bit2 = extractBit (idQubit2, index);
    if (bit1 && bit2) {
        stateVecReal [index] = - stateVecReal [index];
        stateVecImag [index] = - stateVecImag [index];
    }
}

void statevec_controlledPhaseFlip(Qureg qureg, const int idQubit1, const int idQubit2)
{
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledPhaseFlipKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, idQubit1, idQubit2);
}

__global__ void statevec_multiControlledPhaseFlipKernel(Qureg qureg, long long int mask)
{
    long long int index;
    long long int stateVecSize;

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=stateVecSize) return;

    if (mask == (mask & index) ){
        stateVecReal [index] = - stateVecReal [index];
        stateVecImag [index] = - stateVecImag [index];
    }
}

void statevec_multiControlledPhaseFlip(Qureg qureg, int *controlQubits, int numControlQubits)
{
    int threadsPerCUDABlock, CUDABlocks;
    long long int mask = getQubitBitMask(controlQubits, numControlQubits);
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_multiControlledPhaseFlipKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, mask);
}

__global__ void statevec_swapQubitAmpsKernel(Qureg qureg, int qb1, int qb2) {

    qreal *reVec = qureg.stateVec.real;
    qreal *imVec = qureg.stateVec.imag;
    
    long long int numTasks = qureg.numAmpsPerChunk >> 2; // each iteration updates 2 amps and skips 2 amps
    long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;
    
    long long int ind00, ind01, ind10;
    qreal re01, re10, im01, im10;
  
    // determine ind00 of |..0..0..>, |..0..1..> and |..1..0..>
    ind00 = insertTwoZeroBits(thisTask, qb1, qb2);
    ind01 = flipBit(ind00, qb1);
    ind10 = flipBit(ind00, qb2);

    // extract statevec amplitudes 
    re01 = reVec[ind01]; im01 = imVec[ind01];
    re10 = reVec[ind10]; im10 = imVec[ind10];

    // swap 01 and 10 amps
    reVec[ind01] = re10; reVec[ind10] = re01;
    imVec[ind01] = im10; imVec[ind10] = im01;
}

void statevec_swapQubitAmpsLocal(Qureg qureg, int qb1, int qb2) 
{
    // stage 1 done!
    // chunkId done!
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>2)/threadsPerCUDABlock);
    statevec_swapQubitAmpsKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, qb1, qb2);
}

__global__ void statevec_hadamardKernel (Qureg qureg, const int targetQubit){
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block

    // ----- temp variables
    qreal   stateRealUp,stateRealLo,                             // storage for previous state values
           stateImagUp,stateImagLo;                             // (used in updates)
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    const long long int numTasks=qureg.numAmpsPerChunk>>1;

    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    // ---------------------------------------------------------------- //
    //            rotate                                                //
    // ---------------------------------------------------------------- //

    //! fix -- no necessary for GPU version
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    qreal recRoot2 = 1.0/sqrt(2.0);

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock   = thisTask / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    // store current state vector values in temp variables
    stateRealUp = stateVecReal[indexUp];
    stateImagUp = stateVecImag[indexUp];

    stateRealLo = stateVecReal[indexLo];
    stateImagLo = stateVecImag[indexLo];

    stateVecReal[indexUp] = recRoot2*(stateRealUp + stateRealLo);
    stateVecImag[indexUp] = recRoot2*(stateImagUp + stateImagLo);

    stateVecReal[indexLo] = recRoot2*(stateRealUp - stateRealLo);
    stateVecImag[indexLo] = recRoot2*(stateImagUp - stateImagLo);
}

void statevec_hadamardLocal(Qureg qureg, const int targetQubit) 
{
    // chunkID done!

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_hadamardKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit);
}

__global__ void statevec_controlledNotKernel(Qureg qureg, const int controlQubit, const int targetQubit)
{
    long long int index;
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    long long int stateVecSize;
    const long long int chunkSize=qureg.numAmpsPerChunk;
    const long long int chunkId=qureg.chunkId;
    int controlBit;

    // ----- temp variables
    qreal   stateRealUp,                             // storage for previous state values
           stateImagUp;                             // (used in updates)
    long long int thisBlock,                                           // current block
         indexUp,indexLo;                                     // current index and corresponding index in lower half block
    sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks

    stateVecSize = qureg.numAmpsPerChunk;
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index>=(stateVecSize>>1)) return;
    thisBlock   = index / sizeHalfBlock;
    indexUp     = thisBlock*sizeBlock + index%sizeHalfBlock;
    indexLo     = indexUp + sizeHalfBlock;

    controlBit = extractBit(controlQubit, indexUp+chunkId*chunkSize);
    if (controlBit){
        stateRealUp = stateVecReal[indexUp];
        stateImagUp = stateVecImag[indexUp];

        stateVecReal[indexUp] = stateVecReal[indexLo];
        stateVecImag[indexUp] = stateVecImag[indexLo];

        stateVecReal[indexLo] = stateRealUp;
        stateVecImag[indexLo] = stateImagUp;
    }
}

void statevec_controlledNotLocal(Qureg qureg, const int controlQubit, const int targetQubit)
{
    // chunkID done!
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk)/threadsPerCUDABlock);
    statevec_controlledNotKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit);
}

__device__ __host__ unsigned int log2Int( unsigned int x )
{
    unsigned int ans = 0 ;
    while( x>>=1 ) ans++;
    return ans ;
}

__device__ void reduceBlock(qreal *arrayIn, qreal *reducedArray, int length){
    int i, l, r;
    int threadMax, maxDepth;
    threadMax = length/2;
    maxDepth = log2Int(length/2);

    for (i=0; i<maxDepth+1; i++){
        if (threadIdx.x<threadMax){
            l = threadIdx.x;
            r = l + threadMax;
            arrayIn[l] = arrayIn[r] + arrayIn[l];
        }
        threadMax = threadMax >> 1;
        __syncthreads(); // optimise -- use warp shuffle instead
    }

    if (threadIdx.x==0) reducedArray[blockIdx.x] = arrayIn[0];
}

__global__ void copySharedReduceBlock(qreal*arrayIn, qreal *reducedArray, int length){
    extern __shared__ qreal tempReductionArray[];
    int blockOffset = blockIdx.x*length;
    tempReductionArray[threadIdx.x*2] = arrayIn[blockOffset + threadIdx.x*2];
    tempReductionArray[threadIdx.x*2+1] = arrayIn[blockOffset + threadIdx.x*2+1];
    __syncthreads();
    reduceBlock(tempReductionArray, reducedArray, length);
}



__global__ void statevec_findProbabilityOfZeroKernel(
        Qureg qureg, const int measureQubit, qreal *reducedArray
) {
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         index;                                               // current index for first half block
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    long long int numTasks=qureg.numAmpsPerChunk>>1;
    // (good for shared memory parallelism)

    extern __shared__ qreal tempReductionArray[];

    // ---------------------------------------------------------------- //
    //            dimensions                                            //
    // ---------------------------------------------------------------- //
    sizeHalfBlock = 1LL << (measureQubit);                       // number of state vector elements to sum,
    // and then the number to skip
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks (pairs of measure and skip entries)

    // ---------------------------------------------------------------- //
    //            find probability                                      //
    // ---------------------------------------------------------------- //

    //
    // --- task-based shared-memory parallel implementation
    //

    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;

    thisBlock = thisTask / sizeHalfBlock;
    index     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;
    qreal realVal, imagVal;
    realVal = stateVecReal[index];
    imagVal = stateVecImag[index];  
    tempReductionArray[threadIdx.x] = realVal*realVal + imagVal*imagVal;
    __syncthreads();

    if (threadIdx.x<blockDim.x/2){
        reduceBlock(tempReductionArray, reducedArray, blockDim.x);
    }
}

qreal statevec_findProbabilityOfZeroLocal(Qureg qureg, const int measureQubit)
{
    // chunk ID done!
    
    long long int numValuesToReduce = qureg.numAmpsPerChunk>>1;
    int valuesPerCUDABlock, numCUDABlocks, sharedMemSize;
    qreal stateProb=0;
    int firstTime=1;
    int maxReducedPerLevel = REDUCE_SHARED_SIZE;

    while(numValuesToReduce>1){ 
        if (numValuesToReduce<maxReducedPerLevel){
            // Need less than one CUDA block to reduce values
            valuesPerCUDABlock = numValuesToReduce;
            numCUDABlocks = 1;
        } else {
            // Use full CUDA blocks, with block size constrained by shared mem usage
            valuesPerCUDABlock = maxReducedPerLevel;
            numCUDABlocks = ceil((qreal)numValuesToReduce/valuesPerCUDABlock);
        }
        sharedMemSize = valuesPerCUDABlock*sizeof(qreal);

        if (firstTime){
            statevec_findProbabilityOfZeroKernel<<<numCUDABlocks, valuesPerCUDABlock, sharedMemSize>>>(
                    qureg, measureQubit, qureg.firstLevelReduction);
            firstTime=0;
        } else {
            hipDeviceSynchronize();    
            copySharedReduceBlock<<<numCUDABlocks, valuesPerCUDABlock/2, sharedMemSize>>>(
                    qureg.firstLevelReduction, 
                    qureg.secondLevelReduction, valuesPerCUDABlock); 
            hipDeviceSynchronize();    
            swapDouble(&(qureg.firstLevelReduction), &(qureg.secondLevelReduction));
        }
        numValuesToReduce = numValuesToReduce/maxReducedPerLevel;
    }
    hipMemcpy(&stateProb, qureg.firstLevelReduction, sizeof(qreal), hipMemcpyDeviceToHost);
    return stateProb;
}

/** computes either a real or imag term in the inner product */
__global__ void statevec_calcInnerProductKernel(
    int getRealComp,
    qreal* vecReal1, qreal* vecImag1, qreal* vecReal2, qreal* vecImag2, 
    long long int numTermsToSum, qreal* reducedArray) 
{
    long long int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= numTermsToSum) return;
    
    // choose whether to calculate the real or imaginary term of the inner product
    qreal innerProdTerm;
    if (getRealComp)
        innerProdTerm = vecReal1[index]*vecReal2[index] + vecImag1[index]*vecImag2[index];
    else
        innerProdTerm = vecReal1[index]*vecImag2[index] - vecImag1[index]*vecReal2[index];
    
    // array of each thread's collected sum term, to be summed
    extern __shared__ qreal tempReductionArray[];
    tempReductionArray[threadIdx.x] = innerProdTerm;
    __syncthreads();
    
    // every second thread reduces
    if (threadIdx.x<blockDim.x/2)
        reduceBlock(tempReductionArray, reducedArray, blockDim.x);
}

/** Terrible code which unnecessarily individually computes and sums the real and imaginary components of the
 * inner product, so as to not have to worry about keeping the sums separated during reduction.
 * Truly disgusting, probably doubles runtime, please fix.
 * @TODO could even do the kernel twice, storing real in bra.reduc and imag in ket.reduc?
 */
Complex statevec_calcInnerProductLocal(Qureg bra, Qureg ket) {
    // chunkID done!

    qreal innerProdReal, innerProdImag;
    
    int getRealComp;
    long long int numValuesToReduce;
    int valuesPerCUDABlock, numCUDABlocks, sharedMemSize;
    int maxReducedPerLevel;
    int firstTime;
    
    // compute real component of inner product
    getRealComp = 1;
    numValuesToReduce = bra.numAmpsPerChunk;
    maxReducedPerLevel = REDUCE_SHARED_SIZE;
    firstTime = 1;
    while (numValuesToReduce > 1) {
        if (numValuesToReduce < maxReducedPerLevel) {
            valuesPerCUDABlock = numValuesToReduce;
            numCUDABlocks = 1;
        }
        else {
            valuesPerCUDABlock = maxReducedPerLevel;
            numCUDABlocks = ceil((qreal)numValuesToReduce/valuesPerCUDABlock);
        }
        sharedMemSize = valuesPerCUDABlock*sizeof(qreal);
        if (firstTime) {
             statevec_calcInnerProductKernel<<<numCUDABlocks, valuesPerCUDABlock, sharedMemSize>>>(
                 getRealComp,
                 bra.stateVec.real, bra.stateVec.imag, 
                 ket.stateVec.real, ket.stateVec.imag, 
                 numValuesToReduce, 
                 bra.firstLevelReduction);
            firstTime = 0;
        } else {
            hipDeviceSynchronize();    
            copySharedReduceBlock<<<numCUDABlocks, valuesPerCUDABlock/2, sharedMemSize>>>(
                    bra.firstLevelReduction, 
                    bra.secondLevelReduction, valuesPerCUDABlock); 
            hipDeviceSynchronize();    
            swapDouble(&(bra.firstLevelReduction), &(bra.secondLevelReduction));
        }
        numValuesToReduce = numValuesToReduce/maxReducedPerLevel;
    }
    hipMemcpy(&innerProdReal, bra.firstLevelReduction, sizeof(qreal), hipMemcpyDeviceToHost);
    
    // compute imag component of inner product
    getRealComp = 0;
    numValuesToReduce = bra.numAmpsPerChunk;
    maxReducedPerLevel = REDUCE_SHARED_SIZE;
    firstTime = 1;
    while (numValuesToReduce > 1) {
        if (numValuesToReduce < maxReducedPerLevel) {
            valuesPerCUDABlock = numValuesToReduce;
            numCUDABlocks = 1;
        }
        else {
            valuesPerCUDABlock = maxReducedPerLevel;
            numCUDABlocks = ceil((qreal)numValuesToReduce/valuesPerCUDABlock);
        }
        sharedMemSize = valuesPerCUDABlock*sizeof(qreal);
        if (firstTime) {
             statevec_calcInnerProductKernel<<<numCUDABlocks, valuesPerCUDABlock, sharedMemSize>>>(
                 getRealComp,
                 bra.stateVec.real, bra.stateVec.imag, 
                 ket.stateVec.real, ket.stateVec.imag, 
                 numValuesToReduce, 
                 bra.firstLevelReduction);
            firstTime = 0;
        } else {
            hipDeviceSynchronize();    
            copySharedReduceBlock<<<numCUDABlocks, valuesPerCUDABlock/2, sharedMemSize>>>(
                    bra.firstLevelReduction, 
                    bra.secondLevelReduction, valuesPerCUDABlock); 
            hipDeviceSynchronize();    
            swapDouble(&(bra.firstLevelReduction), &(bra.secondLevelReduction));
        }
        numValuesToReduce = numValuesToReduce/maxReducedPerLevel;
    }
    hipMemcpy(&innerProdImag, bra.firstLevelReduction, sizeof(qreal), hipMemcpyDeviceToHost);
    
    // return complex
    Complex innerProd;
    innerProd.real = innerProdReal;
    innerProd.imag = innerProdImag;
    return innerProd;
}



__global__ void statevec_collapseToKnownProbOutcomeKernel(Qureg qureg, int measureQubit, int outcome, qreal totalProbability)
{
    // ----- sizes
    long long int sizeBlock,                                           // size of blocks
         sizeHalfBlock;                                       // size of blocks halved
    // ----- indices
    long long int thisBlock,                                           // current block
         index;                                               // current index for first half block
    // ----- measured probability
    qreal   renorm;                                    // probability (returned) value
    // ----- temp variables
    long long int thisTask;                                   // task based approach for expose loop with small granularity
    // (good for shared memory parallelism)
    long long int numTasks=qureg.numAmpsPerChunk>>1;

    // ---------------------------------------------------------------- //
    //            dimensions                                            //
    // ---------------------------------------------------------------- //
    sizeHalfBlock = 1LL << (measureQubit);                       // number of state vector elements to sum,
    // and then the number to skip
    sizeBlock     = 2LL * sizeHalfBlock;                           // size of blocks (pairs of measure and skip entries)

    // ---------------------------------------------------------------- //
    //            find probability                                      //
    // ---------------------------------------------------------------- //

    //
    // --- task-based shared-memory parallel implementation
    //
    renorm=1/sqrt(totalProbability);
    qreal *stateVecReal = qureg.stateVec.real;
    qreal *stateVecImag = qureg.stateVec.imag;

    thisTask = blockIdx.x*blockDim.x + threadIdx.x;
    if (thisTask>=numTasks) return;
    thisBlock = thisTask / sizeHalfBlock;
    index     = thisBlock*sizeBlock + thisTask%sizeHalfBlock;

    if (outcome==0){
        stateVecReal[index]=stateVecReal[index]*renorm;
        stateVecImag[index]=stateVecImag[index]*renorm;

        stateVecReal[index+sizeHalfBlock]=0;
        stateVecImag[index+sizeHalfBlock]=0;
    } else if (outcome==1){
        stateVecReal[index]=0;
        stateVecImag[index]=0;

        stateVecReal[index+sizeHalfBlock]=stateVecReal[index+sizeHalfBlock]*renorm;
        stateVecImag[index+sizeHalfBlock]=stateVecImag[index+sizeHalfBlock]*renorm;
    }
}

/*
 * outcomeProb must accurately be the probability of that qubit outcome in the state-vector, or
 * else the state-vector will lose normalisation
 */
void statevec_collapseToKnownProbOutcomeLocal(Qureg qureg, const int measureQubit, int outcome, qreal outcomeProb)
{
    // stage 1 done!
    // chunkId done!
    
    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
    statevec_collapseToKnownProbOutcomeKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, measureQubit, outcome, outcomeProb);
}



__global__ void statevec_setWeightedQuregKernel(Complex fac1, Qureg qureg1, Complex fac2, Qureg qureg2, Complex facOut, Qureg out) {

    long long int ampInd = blockIdx.x*blockDim.x + threadIdx.x;
    long long int numAmpsToVisit = qureg1.numAmpsPerChunk;
    if (ampInd >= numAmpsToVisit) return;

    qreal *vecRe1 = qureg1.stateVec.real;
    qreal *vecIm1 = qureg1.stateVec.imag;
    qreal *vecRe2 = qureg2.stateVec.real;
    qreal *vecIm2 = qureg2.stateVec.imag;
    qreal *vecReOut = out.stateVec.real;
    qreal *vecImOut = out.stateVec.imag;

    qreal facRe1 = fac1.real; 
    qreal facIm1 = fac1.imag;
    qreal facRe2 = fac2.real;
    qreal facIm2 = fac2.imag;
    qreal facReOut = facOut.real;
    qreal facImOut = facOut.imag;

    qreal re1,im1, re2,im2, reOut,imOut;
    long long int index = ampInd;

    re1 = vecRe1[index]; im1 = vecIm1[index];
    re2 = vecRe2[index]; im2 = vecIm2[index];
    reOut = vecReOut[index];
    imOut = vecImOut[index];

    vecReOut[index] = (facReOut*reOut - facImOut*imOut) + (facRe1*re1 - facIm1*im1) + (facRe2*re2 - facIm2*im2);
    vecImOut[index] = (facReOut*imOut + facImOut*reOut) + (facRe1*im1 + facIm1*re1) + (facRe2*im2 + facIm2*re2);
}

void statevec_setWeightedQureg(Complex fac1, Qureg qureg1, Complex fac2, Qureg qureg2, Complex facOut, Qureg out) {

    long long int numAmpsToVisit = qureg1.numAmpsPerChunk;

    int threadsPerCUDABlock, CUDABlocks;
    threadsPerCUDABlock = DEFAULT_THREADS_PER_BLOCK;
    CUDABlocks = ceil(numAmpsToVisit / (qreal) threadsPerCUDABlock);
    statevec_setWeightedQuregKernel<<<CUDABlocks, threadsPerCUDABlock>>>(
        fac1, qureg1, fac2, qureg2, facOut, out
    );
}

void seedQuESTDefaultLocal(){
    // init MT random number generator with three keys -- time and pid
    // for the MPI version, it is ok that all procs will get the same seed as random numbers will only be 
    // used by the master process

    unsigned long int key[2];
    getQuESTDefaultSeedKey(key); 
    init_by_array(key, 2); 
}  


#ifdef __cplusplus
}
#endif
